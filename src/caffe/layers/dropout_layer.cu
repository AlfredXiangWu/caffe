#include "hip/hip_runtime.h"
#include <algorithm>
#include <limits>
#include <vector>

#include "caffe/common.hpp"
#include "caffe/layer.hpp"
#include "caffe/syncedmem.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

#define DEBUG_WUHAO 1
#include  <iostream>

namespace caffe {


template <typename Dtype>
__global__ void DropoutForward(const int n, const Dtype* in,
    const unsigned int* mask, const unsigned int threshold, const float scale,
    Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] * (mask[index] > threshold) * scale;
  }
}

template <typename Dtype>
void DropoutLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = (*top)[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  if (Caffe::phase() == Caffe::TRAIN) {
    unsigned int* mask =
        static_cast<unsigned int*>(rand_vec_.mutable_gpu_data());
    if (own_mask_) {
      caffe_gpu_rng_uniform(count, mask);
    }

    #if 0 //DEBUG_WUHAO
      const unsigned int* mask_cpu = rand_vec_.cpu_data();
      std::cout << "layer " << this->layer_param_.name();
      std::cout << " own_mask_: " << own_mask_ << std::endl;
      for (int i = 0; i < 100; ++i) {
        std::cout << (Dtype)(mask_cpu[i])/UINT_MAX << " ";
      }
      std::cout << std::endl;
      std::string input_str;
      std::cout << "pause...";
      std::cin >> input_str;
    #endif

    // set thresholds
    // NOLINT_NEXT_LINE(whitespace/operators)
    DropoutForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, mask, uint_thres_, scale_, top_data);
    CUDA_POST_KERNEL_CHECK;
  } else {
    caffe_copy(count, bottom_data, top_data);
  }
}

template <typename Dtype>
__global__ void DropoutBackward(const int n, const Dtype* in_diff,
    const unsigned int* mask, const unsigned int threshold, const float scale,
    Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * scale * (mask[index] > threshold);
  }
}

template <typename Dtype>
void DropoutLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    vector<Blob<Dtype>*>* bottom) {
  if (propagate_down[0]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = (*bottom)[0]->mutable_gpu_diff();
    if (Caffe::phase() == Caffe::TRAIN) {
      const unsigned int* mask =
          static_cast<const unsigned int*>(rand_vec_.gpu_data());
      const int count = (*bottom)[0]->count();
      // NOLINT_NEXT_LINE(whitespace/operators)
      DropoutBackward<Dtype><<<CAFFE_GET_BLOCKS(count),
        CAFFE_CUDA_NUM_THREADS>>>(
          count, top_diff, mask, uint_thres_, scale_, bottom_diff);
      CUDA_POST_KERNEL_CHECK;
    } else {
      caffe_copy(top[0]->count(), top_diff, bottom_diff);
    }
  }
}

INSTANTIATE_CLASS(DropoutLayer);


}  // namespace caffe
