#include <vector>

#include "caffe/blob.hpp"
#include "caffe/layer.hpp"
#include "caffe/proto/caffe.pb.h"

#include "caffe/layers/loss_layer.hpp"
#include "caffe/layers/triplet_loss_layer.hpp"

namespace caffe {

template <typename Dtype>
void TripletLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom, 
    const vector<Blob<Dtype>*>& top) {

    Forward_cpu(bottom, top);
}

template <typename Dtype>
void TripletLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

    Backward_cpu(top, propagate_down, bottom);
}
    
INSTANTIATE_LAYER_GPU_FUNCS(TripletLossLayer);

}