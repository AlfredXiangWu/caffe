#include <vector>

#include "caffe/data_layers.hpp"

namespace caffe {

template <typename Dtype>
void CharSeqDataLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, vector<Blob<Dtype>*>* top) {
  // First, join the thread
  this->JoinPrefetchThread();
  // Copy the data
  caffe_copy(this->prefetch_data_.count(), this->prefetch_data_.cpu_data(),
      (*top)[0]->mutable_gpu_data());

  if (this->output_labels_) {
    caffe_copy(this->prefetch_label_.count(), this->prefetch_label_.cpu_data(),
        (*top)[1]->mutable_gpu_data());
  }

  //(*top)[1]->cpu_data();

  for (int i=0; i < max_length_; i++)
  {
    Dtype* temp_label = (*top)[i+2]->mutable_cpu_data();
    for (int nbatch=0; nbatch < this->layer_param_.char_seq_data_param().batch_size(); nbatch++)
    {
      //this operation is ill-posed, it may be fixed latter. 
      int idx = (int) (*top)[1]->data_at(nbatch, 0, 0, 0);

      *(temp_label + nbatch) = *(character_label_ + idx*max_length_+i);
#if 0//DEBUG
      std::cout << "label1:" << *(character_label_ + nbatch*max_length_+i)<<"\n";
      std::cout << "temp: " << *(temp_label + nbatch) << "\n";
      std::cout << "top: " << *((*top)[i+2]->mutable_cpu_data()+nbatch)<<"\n";
      std::cout << "pause..";
      string str;
      std::cin >> str;
#endif
    }
    (*top)[i+2]->gpu_data();
  }

  this->CreatePrefetchThread();
}

INSTANTIATE_CLASS(CharSeqDataLayer);

}  // namespace caffe
