#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/blob.hpp"
#include "caffe/layer.hpp"
#include "caffe/proto/caffe.pb.h"

#include "caffe/layers/relevance_constraint_layer.hpp"

namespace caffe {

template <typename Dtype>
void RelevanceConstraintLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom, 
    const vector<Blob<Dtype>*>& top) {

    //Forward_cpu(bottom, top);

    const Dtype* bottom_data_1 = bottom[0]->gpu_data();
    const Dtype* bottom_data_2 = bottom[1]->gpu_data();

    Dtype* top_data_1 = top[0]->mutable_gpu_data();
    Dtype* top_data_2 = top[1]->mutable_gpu_data();

    const Dtype* W_1 = this->blobs_[0]->gpu_data();
    const Dtype* W_2 = this->blobs_[1]->gpu_data();

    if (M_ == 1) {
        caffe_gpu_gemv<Dtype>(CblasNoTrans, N_, K_, (Dtype)1.,
                             W_1, bottom_data_1, (Dtype)0., top_data_1);
        caffe_gpu_gemv<Dtype>(CblasNoTrans, N_, K_, (Dtype)1.,
                             W_2, bottom_data_2, (Dtype)0., top_data_2);
        if (bias_term_) {
          caffe_gpu_axpy<Dtype>(N_, bias_multiplier_1_.cpu_data()[0],
                                this->blobs_[2]->gpu_data(), top_data_1);
          caffe_gpu_axpy<Dtype>(N_, bias_multiplier_2_.cpu_data()[0],
                                this->blobs_[3]->gpu_data(), top_data_2);
        }
    } else {
        caffe_gpu_gemm<Dtype>(CblasNoTrans,
                              CblasTrans,
                              M_, N_, K_, (Dtype)1.,
                              bottom_data_1, W_1, (Dtype)0., top_data_1);
        caffe_gpu_gemm<Dtype>(CblasNoTrans,
                              CblasTrans,
                              M_, N_, K_, (Dtype)1.,
                              bottom_data_2, W_2, (Dtype)0., top_data_2);
        if (bias_term_){
          caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, N_, 1, (Dtype)1.,
                                bias_multiplier_1_.gpu_data(),
                                this->blobs_[2]->gpu_data(), (Dtype)1., top_data_1);
          caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, N_, 1, (Dtype)1.,
                                bias_multiplier_2_.gpu_data(),
                                this->blobs_[3]->gpu_data(), (Dtype)1., top_data_2);
        }
    }

    if(lambda_ > 0) {
        // compute sum oof WW^T
        caffe_cpu_gemm<Dtype>(CblasNoTrans, CblasTrans,
            N_, N_, K_,
            (Dtype)1., W_1, W_1,
            (Dtype)0., WW_1.mutable_cpu_data());
        caffe_cpu_gemm<Dtype>(CblasNoTrans, CblasTrans,
            N_, N_, K_,
            (Dtype)1., W_2, W_2,
            (Dtype)0., WW_2.mutable_cpu_data());

        caffe_copy(N_*N_, WW_2.cpu_data(), WW_sum.mutable_cpu_data());
        caffe_axpy(N_*N_, (Dtype)1., WW_1.cpu_data(), WW_sum.mutable_cpu_data());

        // SVD
        int info = caffe_cpu_gesvd<Dtype>(N_, N_, WW_sum.mutable_cpu_data(),  S.mutable_cpu_data(), 
                    U.mutable_cpu_data(),  VT.mutable_cpu_data());

        //caffe_sqrt<Dtype>(N_, S.cpu_data(), S.mutable_cpu_data());

        // compute inverse of gamma
        Dtype* diag_S_data = diag_S.mutable_cpu_data();
        Dtype* inverse_gamma_data = inverse_gamma_.mutable_cpu_data();
        caffe_set(N_*N_, Dtype(0), diag_S_data);
        for(int i = 0; i < N_; ++i) {
            diag_S_data[i * N_ + i] = sqrt(S.cpu_data()[i]);
        }

        caffe_cpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, N_, N_, N_, (Dtype)1., U.cpu_data(), 
            diag_S.cpu_data(), (Dtype)0., inverse_gamma_data);
        caffe_cpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, N_, N_, N_, (Dtype)1., inverse_gamma_.cpu_data(), 
            VT.cpu_data(), (Dtype)0., inverse_gamma_data);

        Dtype trace = caffe_cpu_asum<Dtype>(N_, diag_S.cpu_data());
        for(int i = 0; i < N_; ++i) {
            for(int j = 0; j < N_; ++j) {
                inverse_gamma_data[i * N_ + j] = inverse_gamma_.cpu_data()[i] / (trace + 1e-5);
            }
        }

        info = caffe_cpu_getri<Dtype>(N_, inverse_gamma_data);
    }
}

template <typename Dtype>
void RelevanceConstraintLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

    //Backward_cpu(top, propagate_down, bottom);
    const Dtype* W_1 = this->blobs_[0]->gpu_data();
    const Dtype* W_2 = this->blobs_[1]->gpu_data();
    const Dtype* inverse_gamma_data = inverse_gamma_.cpu_data();

    const Dtype* bottom_data_1 = bottom[0]->gpu_data();
    const Dtype* bottom_data_2 = bottom[1]->gpu_data();

    //update W
    if (this->param_propagate_down_[0]) {
        const Dtype* top_diff_1 = top[0]->gpu_diff();
        // Gradient with respect to weight
        caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans,
          N_, K_, M_,
          (Dtype)1., top_diff_1, bottom_data_1,
          (Dtype)1., this->blobs_[0]->mutable_gpu_diff());

        if(lambda_ > 0) {
            caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans,
                N_, K_, N_,
                lambda_, inverse_gamma_data, W_1,
                (Dtype)1., this->blobs_[0]->mutable_gpu_diff());
        }
    }

    if (this->param_propagate_down_[1]) {
        const Dtype* top_diff_2 = top[1]->gpu_diff();
        // Gradient with respect to weight
        caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans,
          N_, K_, M_,
          (Dtype)1., top_diff_2, bottom_data_2,
          (Dtype)1., this->blobs_[1]->mutable_gpu_diff());

        if(lambda_ > 0) {
            caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans,
                N_, K_, N_,
                lambda_, inverse_gamma_data, W_2,
                (Dtype)1., this->blobs_[1]->mutable_gpu_diff());
        }
    }

    // update b
    if (bias_term_ && this->param_propagate_down_[2]) {
        const Dtype* top_diff_1 = top[0]->gpu_diff();
        // Gradient with respect to bias
        caffe_gpu_gemv<Dtype>(CblasTrans, M_, N_, (Dtype)1., top_diff_1,
            bias_multiplier_1_.gpu_data(), (Dtype)1.,
            this->blobs_[2]->mutable_gpu_diff());
    }

    if (bias_term_ && this->param_propagate_down_[3]) {
        const Dtype* top_diff_2 = top[1]->gpu_diff();
        // Gradient with respect to bias
        caffe_gpu_gemv<Dtype>(CblasTrans, M_, N_, (Dtype)1., top_diff_2,
            bias_multiplier_2_.gpu_data(), (Dtype)1.,
            this->blobs_[3]->mutable_gpu_diff());
    }

    // propagation data
    if (propagate_down[0]) {
        const Dtype* top_diff_1 = top[0]->gpu_diff();
        // Gradient with respect to bottom data
        caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans,
            M_, K_, N_,
            (Dtype)1., top_diff_1, this->blobs_[0]->gpu_data(),
            (Dtype)0., bottom[0]->mutable_gpu_diff());
    }

    if (propagate_down[1]) {
        const Dtype* top_diff_2 = top[1]->gpu_diff();
        // Gradient with respect to bottom data
        caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans,
            M_, K_, N_,
            (Dtype)1., top_diff_2, this->blobs_[1]->gpu_data(),
            (Dtype)0., bottom[1]->mutable_gpu_diff());
    }

}
    
INSTANTIATE_LAYER_GPU_FUNCS(RelevanceConstraintLayer);

}