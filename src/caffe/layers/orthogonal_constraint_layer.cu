#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layers/orthogonal_constraint_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void OrthogonalConstraintLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype *top_data = top[0]->mutable_gpu_data();
    const Dtype* weight = this->blobs_[0]->gpu_data();

    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans,
        M_, N_, K_, (Dtype)1.,
        bottom_data, weight, (Dtype)0., top_data);

    if (bias_term_) {
        caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, N_, 1, (Dtype)1.,
            bias_multiplier_.gpu_data(),
            this->blobs_[1]->gpu_data(), (Dtype)1., top_data);
    }

    if(lambda_ > 0) {
        caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans,
            N_, N_, K_,
            (Dtype)1., weight, weight,
            (Dtype)0., WW_.mutable_gpu_data());
        caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, 
            N_, K_, N_, 
            (Dtype)1., WW_.gpu_data(), weight, 
            (Dtype)0., W_gradient_.mutable_gpu_data());
        caffe_gpu_axpby<Dtype>(N_*K_, (Dtype)(-1), weight, 
            (Dtype)1., W_gradient_.mutable_gpu_data());
    }
}

template <typename Dtype>
void OrthogonalConstraintLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
    if (this->param_propagate_down_[0]) { 
        const Dtype* top_diff = top[0]->gpu_diff();
        const Dtype* bottom_data = bottom[0]->gpu_data();
        caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans,
            N_, K_, M_,
            (Dtype)1., top_diff, bottom_data,
            (Dtype)0., this->blobs_[0]->mutable_gpu_diff());

        if(lambda_ > 0) {
            caffe_gpu_axpby(N_*K_, lambda_, W_gradient_.gpu_data(), 
                (Dtype)1., this->blobs_[0]->mutable_gpu_diff());
        }
    }

    if (bias_term_ && this->param_propagate_down_[1]) {
        const Dtype* top_diff = top[0]->gpu_diff();
        // Gradient with respect to bias
        caffe_gpu_gemv<Dtype>(CblasTrans, M_, N_, (Dtype)1., top_diff,
            bias_multiplier_.gpu_data(), (Dtype)1.,
            this->blobs_[1]->mutable_gpu_diff());
    }

    if (propagate_down[0]) {
        const Dtype* top_diff = top[0]->gpu_diff();
        // Gradient with respect to bottom data
        caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans,
            M_, K_, N_,
            (Dtype)1., top_diff, this->blobs_[0]->gpu_data(),
            (Dtype)0., bottom[0]->mutable_gpu_diff());
    }
    
}


INSTANTIATE_LAYER_GPU_FUNCS(OrthogonalConstraintLayer);

} // namespace caffe